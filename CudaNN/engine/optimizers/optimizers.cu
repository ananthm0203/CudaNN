#include "optimizers.h"
#include "../utils/cuda_utils.cuh"
#include "../utils/general_kernels.cuh"

#include <hip/hip_runtime.h>

void Optimizer::update_grads()
{

	auto f(static_cast<std::function<void(size_t, float*, float*)>>(
		[&](size_t ind, float* A, float* B) { B[ind] += A[ind] / batch_size; }
	));

	for (auto& wgp : weights)
	{
		float* grad_d;
		float* accum_grad_d;

		checkCuda(hipMalloc(&grad_d, wgp.first->get_shape().size));
		checkCuda(hipMalloc(&accum_grad_d, wgp.first->get_shape().size));

		checkCuda(hipMemcpy(grad_d, wgp.first->gradient().raw(), wgp.first->get_shape().size, hipMemcpyHostToDevice));
		checkCuda(hipMemcpy(accum_grad_d, wgp.second.raw(), wgp.first->get_shape().size, hipMemcpyHostToDevice));

		auto& shape = wgp.first->get_shape();

		DEFAULT_CUDA_DIMS_FROM_SHAPE(shape);

		elemwiseOpKernel << <blocksPerGrid, threadsPerBlock >> > (shape.H, shape.W, shape.C, f, grad_d, accum_grad_d);

		checkCuda(hipFree(accum_grad_d));
		checkCuda(hipFree(grad_d));
	}
}
