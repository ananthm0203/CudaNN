#include "hip/hip_runtime.h"
#include "adam.h"
#include "../optimizers.h"
#include "../../utils/cuda_utils.cuh"
#include "../../utils/general_kernels.cuh"

__global__ void Adam::update()
{
	for (size_t i = 0; i < weights.size(); ++i)
	{
		auto& wgp = weights[i];
		auto& mvp = mv_pairs[i];

		auto& weight = wgp.first;
		auto& grad = wgp.second;
		auto& shape = grad.get_shape();

		float* grad_d;
		float* M_d;
		float* V_d;

		checkCuda(hipMalloc(&grad_d, shape.size));
		checkCuda(hipMalloc(&M_d, shape.size));
		checkCuda(hipMalloc(&V_d, shape.size));

		checkCuda(hipMemcpy(grad_d, grad.raw(), shape.size, hipMemcpyHostToDevice));
		checkCuda(hipMemcpy(M_d, mvp.first.raw(), shape.size, hipMemcpyHostToDevice));
		checkCuda(hipMemcpy(V_d, mvp.second.raw(), shape.size, hipMemcpyHostToDevice));

		DEFAULT_CUDA_DIMS_FROM_SHAPE(shape);

		auto f1(static_cast<std::function<void(size_t, float*, float*)>>(
			[&](size_t ind, float* M_d, float* grad) { M_d[ind] = beta1 * M_d[ind] + (1 - beta1) * grad[ind]; }
		));
		elemwiseOpKernel << <blocksPerGrid, threadsPerBlock >> > (shape.H, shape.W, shape.C, f1, M_d, grad_d);

		auto f2(static_cast<std::function<void(size_t, float*, float*)>>(
			[&](size_t ind, float* V_d, float* grad) { V_d[ind] = beta2 * V_d[ind] + (1 - beta2) * grad[ind] * grad[ind]; }
		));
		elemwiseOpKernel << <blocksPerGrid, threadsPerBlock >> > (shape.H, shape.W, shape.C, f2, V_d, grad_d);

		float* weight_d = grad_d;

		checkCuda(hipMemcpy(weight_d, weight, shape.size, hipMemcpyHostToDevice));

		auto f3(static_cast<std::function<void(size_t, float*, float*, float*)>>(
			[&](size_t ind, float* M_d, float* V_d, float* W_d)
			{
				float beta1_t = std::powf(beta1, timestep);
		float beta2_t = std::powf(beta2, timestep);
		float M_hat_d = M_d[ind] / (1 - beta1_t);
		float V_hat_d = V_d[ind] / (1 - beta2_t);
		W_d[ind] -= (lr * M_hat_d) / (std::sqrtf(V_hat_d) + epsilon);
			}
		));
		elemwiseOpKernel << <blocksPerGrid, threadsPerBlock >> > (shape.H, shape.W, shape.C, f3, M_d, V_d, weight_d);

		checkCuda(hipMemcpy(weight->raw(), weight_d, shape.size, hipMemcpyDeviceToHost));
		checkCuda(hipMemcpy(mvp.first.raw(), M_d, shape.size, hipMemcpyDeviceToHost));
		checkCuda(hipMemcpy(mvp.second.raw(), V_d, shape.size, hipMemcpyDeviceToHost));

		checkCuda(hipFree(weight_d));
		checkCuda(hipFree(M_d));
		checkCuda(hipFree(V_d));
	}
}