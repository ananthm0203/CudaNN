#ifndef KERNEL_CUH
#define KERNEL_CUH

#include "../../utils/cuda_utils.cuh"
#include "../../activations/activations.h"
#include "dense.h"

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <stdint.h>
#include <memory>
#include <vector>
#include <string>

void denseForward(
	size_t C_in,
	size_t C_out,
	float* X, float* W, float* B, float* Z, float* R, Activation* activation)
{
	dim3 blocksPerGrid(1, TILENO(C_out));
	dim3 threadsPerBlock(1, TILE_WIDTH);
	size_t X_memsize = C_in * sizeof(float);
	size_t W_memsize = C_in * C_out * sizeof(float);
	size_t R_memsize = C_out * sizeof(float);

	// Allocate memory on the device
	float* X_d, * W_d, * R_d;
	checkCuda(hipMalloc(&X_d, X_memsize));
	checkCuda(hipMalloc(&W_d, W_memsize));
	checkCuda(hipMalloc(&R_d, R_memsize));

	// Copy from the host to the device
	checkCuda(hipMemcpy(X_d, X, X_memsize, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(W_d, W, W_memsize, hipMemcpyHostToDevice));

	// Perform matrix multiplication on the device
	matMulKernel << <blocksPerGrid, threadsPerBlock >> > (W_d, X_d, R_d, C_out, C_in, 1, 1);

	// Free memory on the device
	checkCuda(hipFree(W_d));
	checkCuda(hipFree(X_d));

	// If we need to add bias...
	if (B)
	{
		size_t B_memsize = C_out * sizeof(float);
		// Allocate memory on the device
		float* B_d;
		checkCuda(hipMalloc(&B_d, B_memsize));

		// Copy from the host to the device
		checkCuda(hipMemcpy(B_d, B, B_memsize, hipMemcpyHostToDevice));

		// Perform R (mat.) + Bias (vect.)
		vectMatAddKernel << <blocksPerGrid, threadsPerBlock >> > (B_d, R_d, R_d, C_out, 1, 1);

		// Free memory on the device
		checkCuda(hipFree(B_d));
	}

	if (activation)
	{
		checkCuda(hipMemcpy(Z, R_d, R_memsize, hipMemcpyDeviceToHost));
		activation->forward(R_d);
	}

	// Copy back to device
	checkCuda(hipMemcpy(R, R_d, R_memsize, hipMemcpyDeviceToHost));

	// Free result matrix
	checkCuda(hipFree(R_d));
}

void denseBackprop(
	size_t C_in,
	size_t C_out,
	float* grad,
	float* new_grad,
	float* z,
	float* X, 
	float* W, 
	float* B, 
	float* W_grad, 
	float* B_grad, 
	Activation* activation)
{
	float* grad_d = nullptr;
	size_t grad_memsize = C_in * sizeof(float);
	if (activation)
	{
		if (activation->in_place_grad())
		{
			float* z_d;
			checkCuda(hipMalloc(&z_d, grad_memsize));
			checkCuda(hipMemcpy(z_d, z, grad_memsize, hipMemcpyHostToDevice));
			activation->backprop(z_d, nullptr);
			checkCuda(hipMalloc(&grad_d, grad_memsize));
			checkCuda(hipMemcpy(grad_d, grad, grad_memsize, hipMemcpyHostToDevice));
			dim3 blocksPerGrid(TILENO(C_out));
			dim3 threadsPerBlock(TILE_WIDTH);
			elemwiseOpKernel<MulOp> << <blocksPerGrid, threadsPerBlock >> > (grad_d, z_d, grad_d,
				C_out, 1, 1);
			checkCuda(hipFree(z_d));
		}
		else
		{
			float* z_d;
			float* grad_z_d;
			float* new_grad_d;
			auto grad_z_memsize = C_out * C_out * sizeof(float); // This has already been asserted in operator()
			checkCuda(hipMalloc(&z_d, grad_memsize));
			checkCuda(hipMemcpy(z_d, z, grad_memsize, hipMemcpyHostToDevice));
			checkCuda(hipMalloc(&grad_z_d, grad_z_memsize));
			activation->backprop(z_d, grad_z_d);
			checkCuda(hipFree(z_d));
			checkCuda(hipMalloc(&new_grad_d, grad_memsize));
			checkCuda(hipMalloc(&grad_d, grad_memsize));
			checkCuda(hipMemcpy(grad_d, grad, grad_memsize, hipMemcpyHostToDevice));
			dim3 blocksPerGrid(TILENO(C_out), TILENO(C_out));
			dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
			matMulKernel << <blocksPerGrid, threadsPerBlock >> > (grad_z_d, grad_d, new_grad_d,
				C_out, C_out, 1, 1);
			checkCuda(hipFree(grad_z_d));
			checkCuda(hipFree(grad_d));
			grad_d = new_grad_d;
		}
	}
	else
	{
		checkCuda(hipMalloc(&grad_d, grad_memsize));
		// Copy memory from host to device
		checkCuda(hipMemcpy(grad_d, grad, grad_memsize, hipMemcpyHostToDevice));
	}

	//// Calculate new_grad ////

	// Allocate memory
	auto W_memsize = C_out * C_in * sizeof(float);
	float* W_d;

	// Calculate transpose of the weights
	checkCuda(hipMalloc(&W_d, W_memsize));

	// Copy weights to device
	checkCuda(hipMemcpy(W_d, W, W_memsize, hipMemcpyHostToDevice));

	dim3 blocksPerGrid(TILENO(C_in), TILENO(C_out));
	dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);

	// Allocate memory
	auto new_grad_memsize = C_in * sizeof(float);
	float* new_grad_d;
	checkCuda(hipMalloc(&new_grad_d, new_grad_memsize));

	// Calculate new_grad
	matMulKernel << <blocksPerGrid, threadsPerBlock >> > (grad_d, W_d, new_grad_d, 1, C_out, C_in, 1);

	// Copy new_grad back to host
	checkCuda(hipMemcpy(new_grad, new_grad_d, new_grad_memsize, hipMemcpyDeviceToHost));

	// Free unneeded vars from device memory
	checkCuda(hipFree(new_grad_d));
	checkCuda(hipFree(W_d));

	//// End new_grad calculation ////

	//// Calculate weight gradient ////

	// Calculate the transpose of the input
	auto X_memsize = C_in * sizeof(float);
	float* X_d;
	checkCuda(hipMalloc(&X_d, X_memsize));
	checkCuda(hipMemcpy(X_d, X, X_memsize, hipMemcpyHostToDevice));

	// Allocate memory for the weight gradient
	float* W_grad_d;
	checkCuda(hipMalloc(&W_grad_d, W_memsize));

	// Calculate the weight gradient
	matMulKernel << <blocksPerGrid, threadsPerBlock >> > (grad_d, X_T, W_grad_d, C_out, 1, C_in, 1);

	// Copy back to host
	checkCuda(hipMemcpy(W_grad, W_grad_d, W_memsize, hipMemcpyDeviceToHost));

	//// Calculate bias gradient ////
	// If necessary...
	if (B)
	{
		auto B_memsize = C_out * sizeof(float);
		checkCuda(hipMemcpy(B_grad, grad_d, B_memsize, hipMemcpyDeviceToHost));
	}

	// Free all the memory on the device
	checkCuda(hipFree(W_grad_d));
	checkCuda(hipFree(X_d));
	checkCuda(hipFree(grad_d));
}


#endif // KERNEL_CUH
