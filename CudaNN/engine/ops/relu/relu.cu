#include "hip/hip_runtime.h"
#include "relu.h"
#include "../../utils/general_kernels.cuh"

#include <hip/hip_runtime.h>
#include <functional>
#include <cmath>

void Relu::forwards()
{
	float* relu_d;

	checkCuda(hipMalloc(&relu_d, in->get_shape().size));

	checkCuda(hipMemcpy(relu_d, in->raw(), in->get_shape().size, hipMemcpyHostToDevice));

	auto f(static_cast<std::function<void(size_t, float*)>>(
		[](size_t idx, float* relu_d) { relu_d[idx] = std::max(0.0f, relu_d[idx]); }
	));
	elemwiseOpKernel(in->get_shape().H, in->get_shape().W, in->get_shape().W, f, relu_d);

	checkCuda(hipMemcpy(out.raw(), relu_d, in->get_shape().size, hipMemcpyDeviceToHost));

	checkCuda(hipFree(relu_d));
}

void Relu::backwards()
{
	float* relu_grad_d;

	checkCuda(hipMalloc(&relu_grad_d, in->get_shape().size));

	checkCuda(hipMemcpy(relu_grad_d, in->raw(), in->get_shape().size, hipMemcpyHostToDevice));

	auto f(static_cast<std::function<void(size_t, float*)>>(
		[](size_t idx, float* relu_grad_d) { relu_grad_d[idx] = relu_grad_d[idx] > 0.0f ? 1.0f : 0.0f; }
	));
	elemwiseOpKernel(in->get_shape().H, in->get_shape().W, in->get_shape().W, f, relu_grad_d);

	in->update_gradient(this, relu_grad_d);

	checkCuda(hipFree(relu_grad_d));
}
