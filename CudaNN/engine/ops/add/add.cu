#include "add.h"

void Add::forward()
{
	auto& shape = out.get_shape();

	float* out_d;
	float* in_d;

	checkCuda(hipMalloc(&out_d, shape.size));
	checkCuda(hipMalloc(&in_d, shape.size));

	checkCuda(hipMemcpy(out_d, ins[0]->raw(), shape.size, hipMemcpyHostToDevice));

	DEFAULT_CUDA_DIMS_FROM_SHAPE(shape);

	for (size_t i = 1; i < ins.size(); ++i)
	{
		hipMemcpy(in_d, ins[i]->raw(), shape.size, hipMemcpyHostToDevice);
		elemwiseOpKernel << <blocksPerGrid, threadsPerBlock >> > (shape.H, shape.W, shape.C,
			std::function<void(size_t, float*, float*)>(
				[](size_t ind, float* G1, float* G2)
				{
					G1[ind] += G2[ind];
				}
				),
			out_d, in_d);
	}

	checkCuda(hipFree(in_d));
	checkCuda(hipFree(out_d));
}
