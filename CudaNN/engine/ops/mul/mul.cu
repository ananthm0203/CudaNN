#include "mul.h"
#include "../../utils/general_kernels.cuh"
#include "../../utils/cuda_utils.cuh"

#include <hip/hip_runtime.h>

void MMul::forwards()
{
	float* out_d, * lhs_d, * rhs_d;

	checkCuda(hipMalloc(&out_d, out.get_shape().size));
	checkCuda(hipMalloc(&lhs_d, lhs->get_shape().size));
	checkCuda(hipMalloc(&rhs_d, rhs->get_shape().size));

	checkCuda(hipMemcpy(out_d, out.raw(), out.get_shape().size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(lhs_d, lhs->raw(), lhs->get_shape().size, hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(rhs_d, rhs->raw(), rhs->get_shape().size, hipMemcpyHostToDevice));

	auto& shape = out.get_shape();

	dim3 blocksPerGrid(CEILDIV(shape.H, BLCK_X), CEILDIV(shape.W, BLCK_Y), CEILDIV(shape.W, BLCK_Z));
	dim3 threadsPerBlock(BLCK_X, BLCK_Y, BLCK_Z);

	matMulKernel << <blocksPerGrid, threadsPerBlock >> > (lhs_d, rhs_d, out_d,
		shape.H, lhs->get_shape().H, shape.W, shape.C);

	checkCuda(hipMemcpy(out.raw(), out_d, shape.size, hipMemcpyDeviceToHost));

	checkCuda(hipFree(out_d));
	checkCuda(hipFree(lhs_d));
	checkCuda(hipFree(rhs_d));
}

void MMul::backwards()
{

	if (!lhs->from() && !rhs->from())
	{
		return;
	}

	float* grad_d;

	auto& grad = out.gradient();

	checkCuda(hipMalloc(&grad_d, grad.get_shape().size));
	checkCuda(hipMemcpy(grad_d, grad.raw(), grad.get_shape().size, hipMemcpyHostToDevice));
	
	// Calc LHS grad
	if (lhs->from())
	{
		float* rhs_t_d, * rhs_d;

		checkCuda(hipMalloc(&rhs_d, rhs->get_shape().size));
		checkCuda(hipMalloc(&rhs_t_d, rhs->get_shape().size));

		checkCuda(hipMemcpy(rhs_d, rhs->raw(), rhs->get_shape().size, hipMemcpyHostToDevice));

		{
			DEFAULT_CUDA_DIMS_FROM_SHAPE(rhs->get_shape());
			transposeKernel << <blocksPerGrid, threadsPerBlock >> > (rhs_d, rhs_t_d,
				lhs->get_shape().H, lhs->get_shape().W, lhs->get_shape().C);

			checkCuda(hipFree(rhs_d));
		}


		float* grad_lhs_d;

		checkCuda(hipMalloc(&grad_lhs_d, lhs->get_shape().size));

		{
			DEFAULT_CUDA_DIMS_FROM_SHAPE(lhs->get_shape());
			matMulKernel << <blocksPerGrid, threadsPerBlock >> > (grad_d, rhs_t_d, grad_lhs_d,
				lhs->get_shape().H, rhs->get_shape().W, lhs->get_shape().W, lhs->get_shape().C);
		}

		checkCuda(hipFree(rhs_t_d));

		lhs->update_gradient(this, grad_lhs_d);

		checkCuda(hipFree(grad_lhs_d));
	}

	// Calc RHS grad
	if (rhs->from())
	{
		float* lhs_d, * lhs_t_d;

		checkCuda(hipMalloc(&lhs_d, lhs->get_shape().size));
		checkCuda(hipMalloc(&lhs_t_d, grad.get_shape().size));

		checkCuda(hipMemcpy(lhs_d, lhs->raw(), lhs->get_shape().size, hipMemcpyHostToDevice));

		{
			DEFAULT_CUDA_DIMS_FROM_SHAPE(lhs->get_shape());
			transposeKernel << <blocksPerGrid, threadsPerBlock >> > (lhs_d, lhs_t_d,
				lhs->get_shape().H, lhs->get_shape().W, lhs->get_shape().C);
		}

		checkCuda(hipFree(lhs_d));

		float* grad_rhs_d;

		checkCuda(hipMalloc(&grad_rhs_d, rhs->get_shape().size));

		{
			DEFAULT_CUDA_DIMS_FROM_SHAPE(rhs->get_shape());
			matMulKernel << <blocksPerGrid, threadsPerBlock >> > (lhs_t_d, grad_d, grad_rhs_d,
				rhs->get_shape().H, lhs->get_shape().H, rhs->get_shape().W, rhs->get_shape().C);
		}

		checkCuda(hipFree(lhs_t_d));

		rhs->update_gradient(this, grad_rhs_d);

		checkCuda(hipFree(grad_rhs_d));
	}
	
	checkCuda(hipFree(grad_d));
}
